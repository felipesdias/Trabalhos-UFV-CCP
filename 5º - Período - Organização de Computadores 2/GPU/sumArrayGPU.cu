
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__
void sumArrayGPU(int n, float *I, float *R, float *C, int vet) {

  int i = blockIdx.x * blockDim.x + threadIdx.x; 

  if(vet == 1)
  	i = I[i];
  else
    i = R[i];

  if (i < n)
    C[i] = I[i] + R[i];

}

int main(int argc, char const *argv[]) {

  srand((unsigned) time(NULL));	

  long N = 1 << (argc > 1 ? atol(argv[1]) : 20);
  int block_aux = argc > 2 ? atoi(argv[2]) : 512;
  int unroll = argc > 3 ? atoi(argv[3]) : 1;
  int vet = argc > 4 ? atoi(argv[4]) : 1;

  int n = N / sizeof(float);

std::cout << N << std::endl << n << std::endl << std::endl;

  float *I, *R, *C, *d_I, *d_R, *d_C;

  I = new float[n];
  R = new float[n];
  C = new float[n];
  
  hipMalloc(&d_I, N);
  hipMalloc(&d_R, N);
  hipMalloc(&d_C, N);

  for (int i = 0; i < n; i++) {
    I[i] = i + 0.0f;
    R[i] = (float)( rand() & 0xFF ) / 10.0f;
	C[i] = 0;
  }

  hipMemcpy(d_I, I, N, hipMemcpyHostToDevice);
  hipMemcpy(d_R, R, N, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, N, hipMemcpyHostToDevice);

  dim3 block (block_aux);
  dim3 grid  ((n + block.x - 1) / block.x);

  sumArrayGPU<<<grid, block>>>(n, d_I, d_R, d_C, vet);

  hipMemcpy(C, d_C, N, hipMemcpyDeviceToHost);

  printf("Soma executada com sucesso");

}
